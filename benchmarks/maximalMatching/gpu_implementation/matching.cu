#include "hip/hip_runtime.h"
// This code is part of the Problem Based Benchmark Suite (PBBS)
// Copyright (c) 2011 Guy Blelloch and the PBBS team
//
// Permission is hereby granted, free of charge, to any person obtaining a
// copy of this software and associated documentation files (the
// "Software"), to deal in the Software without restriction, including
// without limitation the rights (to use, copy, modify, merge, publish,
// distribute, sublicense, and/or sell copies of the Software, and to
// permit persons to whom the Software is furnished to do so, subject to
// the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
// OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
// MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
// LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
// OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
// WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#define NOTMAIN 1

#include <iostream>
#include "parlay/parallel.h"
#include "parlay/primitives.h"
#include "common/graph.h"
#include "common/speculative_for.h"
#include "common/get_time.h"
#include "matching.h"

#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main2(void) {
    int N = 1 << 20;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}


using namespace std;

using reservation = pbbs::reservation<edgeId>;

struct matchStep {
    edges const &E;
    parlay::sequence<reservation> &R;
    parlay::sequence<bool> &matched;

    matchStep(edges const &E,
              parlay::sequence<reservation> &R,
              parlay::sequence<bool> &matched)
            : E(E), R(R), matched(matched) {}

    bool reserve(edgeId i) {
        size_t u = E[i].u;
        size_t v = E[i].v;
        if (matched[u] || matched[v] || (u == v)) return 0;
        R[u].reserve(i);
        R[v].reserve(i);
        return 1;
    }

    bool commit(edgeId i) {
        size_t u = E[i].u;
        size_t v = E[i].v;
        if (R[v].check(i)) {
            R[v].reset();
            if (R[u].check(i)) {
                matched[u] = matched[v] = 1;
                return 1;
            }
        } else if (R[u].check(i)) R[u].reset();
        return 0;
    }
};

parlay::sequence<edgeId> maximalMatching(edges const &E) {
//    printf("CUDA starting...");
    main2();
//    printf("CUDA end.");
    size_t n = max(E.numCols, E.numRows);
    size_t m = E.nonZeros;
    timer t("max matching", true);

    parlay::sequence<reservation> R(n);
    parlay::sequence<bool> matched(n, false);
    matchStep mStep(E, R, matched);
    t.next("init");
    pbbs::speculative_for<edgeId>(mStep, 0, m, 10, 0);
    t.next("speculative for");
    parlay::sequence<edgeId> matchingIdx =
            parlay::pack(parlay::delayed_seq<edgeId>(n, [&](size_t i) { return R[i].get(); }),
                         parlay::tabulate(n, [&](size_t i) -> bool { return R[i].reserved(); }));
    t.next("speculative for");
    cout << "number of matches = " << matchingIdx.size() << endl;
    return matchingIdx;
}  
